
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <ctime>
#include <cfloat>
#include <omp.h>
#include <sys/time.h>
#include <cmath>

#define MAX_THREADS 6

struct dense_mtx
{
    uint32_t nrow; /* Number of rows */
    uint32_t ncol; /* Number of columns */
    double *val;   /* Value of this matrix. */
};

struct info
{
		double val;
		uint32_t pos;
};

void single (struct dense_mtx *A, struct dense_mtx *b);
void L2norm (struct dense_mtx *A, struct dense_mtx *b, struct dense_mtx *_b);
void replicate (struct dense_mtx *A, struct dense_mtx *b, struct dense_mtx *_A, struct dense_mtx *_b);
void cuda (struct dense_mtx *A, struct dense_mtx *b);

int 
main (int argc, char** argv)
{
	uint32_t n = atoi (argv[1]);
	uint32_t total_size = n * (n + 1);

  struct dense_mtx A, A1, A2, b, b1, b2;

	A.nrow = n;
	A.ncol = n;
	b.nrow = n;
	b.ncol = 1;
	A.val = (double *) malloc (total_size * sizeof (double));
	b.val = (double *) malloc (n * sizeof (double));

	A1.nrow = A2.nrow = n;
	A1.ncol = A2.ncol = n;
	b1.nrow = b2.nrow = n;
	b1.ncol = b2.ncol = 1;

	srand48 (time (NULL));

	for (uint32_t i = 0; i < total_size; i++)
		A.val [i] = drand48 ();
	  
	for (uint32_t i = 0; i < n; i++)
		b.val [i] = drand48 ();

	A1.val = (double *) malloc (total_size * sizeof (double));
	b1.val = (double *) malloc (n * sizeof (double));

	replicate (&A, &b, &A1, &b1);

	single (&A1, &b1);
	L2norm (&A, &b, &b1);

	free (A1.val);
	free (b1.val);

	A2.val = (double *) malloc (total_size * sizeof (double));
	b2.val = (double *) malloc (n * sizeof (double));

	replicate (&A, &b, &A2, &b2);
  cuda (&A2, &b2);

	L2norm (&A, &b, &b2);

	free (A2.val);
	free (b2.val);

	free (A.val);
	free (b.val);

	return 0;
}

__global__ void
gaussian (double *A, double *b, uint32_t n, uint32_t cur_pos, double* m_ptr)
{
	uint32_t Block = 16;
  uint32_t tx = threadIdx.x;
	uint32_t ty = threadIdx.y;
	uint32_t Row = Block * blockIdx.y + ty + 1 + cur_pos;
	uint32_t Col = Block * blockIdx.x + tx + 1 + cur_pos;
	
	__shared__ double subA [16][16];
	__shared__ double pivot_Row [16];
	__shared__ double pivot_Col [16];
	__shared__ double m;

	/* Get m. */
	if (tx == ty == 0)
		m = *m_ptr;

	__syncthreads ();

	/* Get subA. */	
	if (Row < n && Col < n)
		{
			subA [ty][tx] = A [Row * n + Col];
		}
	else if (Row < n && Col == n)
	  {
			subA [ty][tx] = b [Row];
	  }
	else
		{
			subA [ty][tx] = 0;
		}
	
	/* Get pivots */
	if (ty == 0)
		{
			uint32_t pRow = Block * blockIdx.y + tx + 1 + cur_pos;
			if (pRow < n)
				{
					pivot_Col [tx] = A [pRow * n + cur_pos];
				}
			else
				{
					pivot_Col [tx] = 0;
				}

			if (Col < n)
				{
					pivot_Row [tx] = A [cur_pos * n + Col];
				}
			else if (Col == n)
				{
					pivot_Row [tx] = b [cur_pos];
				}
			else
				{
					pivot_Row [tx] = 0;
				}
		}

	__syncthreads ();

	/* Do gaussian. */
	subA [ty][tx] = subA [ty][tx] - (pivot_Col [ty] / m) * pivot_Row [tx];

	/* Update real matrix. */
	if (Row < n && Col < n)
		{
			A [Row * n + Col] = subA [ty][tx];
		}
	else if (Row < n && Col == n)
	  {
			b [Row] = subA [ty][tx];
	  }
}

__global__ void
swap (double *A, double *b, uint32_t n, uint32_t cur_pos, uint32_t *max_pos_ptr)
{
	uint32_t Block = 256;
	uint32_t idx = threadIdx.x + Block * blockIdx.x + cur_pos;
	__shared__ uint32_t max_pos;

	if (threadIdx.x == 0)
		max_pos = *max_pos_ptr;

	__syncthreads ();
	double temp;
	if (idx < n)
		{
			temp = A [cur_pos * n + idx];
			A [cur_pos * n + idx] = A [max_pos * n + idx];
			A [max_pos * n + idx] = temp;
		}
	else if (idx == n)
		{
			temp = b [cur_pos];
			b [cur_pos] = b [max_pos];
			b [max_pos] = temp;
		}
}

__global__ void
findmaxloc (double *A, uint32_t n, uint32_t cur_pos, uint32_t *max_Row, double *m, struct info *maxBuf)
{

	extern __shared__ struct info pivotCol [];
	
	uint32_t tx = threadIdx.x;
	uint32_t Block = 256;
	uint32_t threshold = 3000;
	uint32_t loop_count, i = 0, j = 0;
	uint32_t remain = n - cur_pos;
	uint32_t size;

	/* Find local maximum values & positions */
	while (remain > 0)
		{
			/* Check if remain is larger than threshold. */
			if (remain > threshold)
				size = threshold;
			else
				size = remain;
		
			/* Calculate loop count. */
			loop_count = (size + 2 * Block - 1) / (2 * Block);

			/* Fetch data to shared memory. */
			while (loop_count > i)
				{
					uint32_t A_index = (cur_pos + i * Block * 2 + tx * 2 + threshold * j);
					uint32_t P_index = i * Block * 2 + tx * 2;
			
					if (A_index < cur_pos + j * threshold + size)
						{
			    		pivotCol [P_index].val = A [A_index * n + cur_pos];
			    		pivotCol [P_index].pos = A_index;
						}

					if ((A_index + 1) < cur_pos + j * threshold + size)
						{
							pivotCol [P_index + 1].val = A [(A_index + 1) * n + cur_pos];
							pivotCol [P_index + 1].pos = A_index + 1;
						}
					
					i++;
				}

			__syncthreads ();

			uint32_t temp = size;
			/* Find max location. */
			while (size > 1)
				{
					i = 0;
					while (loop_count > i)
						{	
							uint32_t P_index = i * Block * 2 + tx * 2;
							uint32_t pos;
							double val;
							if (P_index < size && P_index + 1 < size)
								{
									if (abs (pivotCol [P_index].val) > abs (pivotCol [P_index + 1].val))
										{
											pos = pivotCol [P_index].pos;
											val = pivotCol [P_index].val;
										}
									else
										{
											pos = pivotCol [P_index + 1].pos;
											val = pivotCol [P_index + 1].val;
										}
								}
							else if (P_index < size)
								{
									pos = pivotCol [P_index].pos;
									val = pivotCol [P_index].val;
								}
							__syncthreads ();
							if (P_index < size)
								{
									pivotCol [P_index / 2].pos = pos;
									pivotCol [P_index / 2].val = val;
								}
							__syncthreads ();
							i++;
						}
					size = (size + 1) / 2;
				}

			/* Store local maximum value. */
			if (tx == 0)
				{
					maxBuf [j].val = pivotCol [0].val;
					maxBuf [j].pos = pivotCol [0].pos;
				}

		  __syncthreads ();
			/* Advance. */
			j++;
			remain -= temp;

		}

	/* Find Global maximum value & its position. */

	loop_count = (j + 2 * Block - 1) / (2 * Block);
	i = 0;
	while (loop_count > i)
		{
			uint32_t P_index = i * Block * 2 + tx * 2;
			
			if (P_index < j)
				{
		  		pivotCol [P_index].val = maxBuf [P_index].val;
		  		pivotCol [P_index].pos = maxBuf [P_index].pos;
				}

			if ((P_index + 1) < j)
				{
					pivotCol [P_index + 1].val = maxBuf [P_index].val;
					pivotCol [P_index + 1].pos = maxBuf [P_index].pos;
				}
		
			i++;
		}

	
	size = j;
	while (size > 1)
		{
			i = 0;
			while (loop_count > i)
				{	
					uint32_t P_index = i * Block * 2 + tx * 2;
					uint32_t pos;
					double val;
					if (P_index < size && P_index + 1 < size)
						{
							if (abs (pivotCol [P_index].val) > abs (pivotCol [P_index + 1].val))
								{
									pos = pivotCol [P_index].pos;
									val = pivotCol [P_index].val;
								}
							else
								{
									pos = pivotCol [P_index + 1].pos;
									val = pivotCol [P_index + 1].val;
								}
						}
					else if (P_index < size)
						{
							pos = pivotCol [P_index].pos;
							val = pivotCol [P_index].val;
						}
					__syncthreads ();
					if (P_index < size)
						{
							pivotCol [P_index / 2].pos = pos;
							pivotCol [P_index / 2].val = val;
						}
					__syncthreads ();
					i++;
				}
			size = (size + 1) / 2;
		}

	if (tx == 0)
		{
			*m = pivotCol [0].val;
			*max_Row = pivotCol [0].pos;
		}
}

void
cuda (struct dense_mtx *A, struct dense_mtx *b)
{
	struct timeval start, end;
	uint32_t Block = 16;
  double *A_val, *b_val;

	std::cout << "cuda start..." << std::endl;
	gettimeofday (&start, NULL);

	uint32_t A_size = A->nrow * A->ncol * sizeof (double);
	uint32_t b_size = b->nrow * b->ncol * sizeof (double);

	hipMalloc (&A_val, A_size);
	hipMalloc (&b_val, b_size);

	hipMemcpy (A_val, A->val, A_size, hipMemcpyHostToDevice);
	hipMemcpy (b_val, b->val, b_size, hipMemcpyHostToDevice);

	uint32_t cur_Col, cur_Row;

	uint32_t *max_Row;
	double *m;

	hipMalloc (&max_Row, sizeof (double));
	hipMalloc (&m, sizeof (double));

	struct info *maxBuf;
	
	hipMalloc (&maxBuf, ((A->nrow + 3000 - 1) / 3000) * sizeof (struct info));

	cur_Col = cur_Row = 0;
	while (cur_Row < (A->nrow - 1))
		{
	    /* Find max location. */
			uint32_t threshold = 3000;
			uint32_t cur = A->nrow - cur_Row;
			uint32_t elements;
			if (cur > threshold)
				elements = threshold;
			else
				elements = cur;
			findmaxloc<<<1, 256, elements * sizeof (struct info) >>>(A_val, A->nrow, cur_Row, max_Row, m, maxBuf);
			
			/* Swap. */
			swap<<<((A->ncol + 1 - cur_Col) + 256 - 1) / 256, 256>>>(A_val, b_val, A->nrow, cur_Row, max_Row);
		 
			/* Gaussian. */
			dim3 dimGrid (((A->nrow - cur_Row) + Block - 1) / Block, ((A->ncol - cur_Col - 1) + Block - 1) / Block, 1);
			dim3 dimBlock (Block, Block, 1);
  		gaussian<<<dimGrid, dimBlock>>>(A_val, b_val, A->nrow, cur_Row, m);

			/* Advance */
			cur_Row++;
			cur_Col++;
		}

	/* Get data. */
	hipMemcpy (A->val, A_val, A_size, hipMemcpyDeviceToHost);
	hipMemcpy (b->val, b_val, b_size, hipMemcpyDeviceToHost);

	hipFree (A_val);
	hipFree (b_val);
	hipFree (m);
	hipFree (max_Row);
	hipFree (maxBuf);

  /* Back substitution. */
	for (int32_t i = A->nrow - 1; i >= 0; i--)
		{
			b->val [i] /= A->val [A->ncol * i + i];
			for (uint32_t j = i + 1; j < A->ncol; j++)
				b->val [i] -= b->val [j] * A->val [A->ncol * i + j] / A->val [A->ncol * i + i];
		}

	gettimeofday (&end, NULL);
	std::cout << "cuda end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;
}

void
single (struct dense_mtx *A, struct dense_mtx *b)
{
	struct timeval start, end;
	uint32_t mrow, srow, scol;
	double max,temp, m;

  std::cout << "single start..." << std::endl;

	gettimeofday (&start, NULL);

	
	srow = scol = mrow = 0;

	while (srow < (A->nrow - 1))
		{
			/* Find max location. */
			max = 0;
			for (uint32_t i = srow; i < A->nrow; i++)
				{
					double val = A->val [A->ncol * i + scol];
					double tval, tmax;

					tval = val > 0 ? val : -val;
					tmax = max > 0 ? max : -max;

					if (tval > tmax)
						{
							max = val;
							mrow = i;
						}
				}

			/* Swap. */
		  for (uint32_t j = scol; j < A->ncol; j++)
			  {
			    temp = A->val [A->ncol * srow + j];
		 	    A->val [A->ncol * srow + j] = A->val [A->ncol * mrow + j];
				  A->val [A->ncol * mrow + j] = temp;
			  }
			temp = b->val [srow];
			b->val [srow] = b->val [mrow];
			b->val [mrow] = temp;

			/* Gaussian elimination. */	
			for (uint32_t i = srow + 1; i < A->nrow; i++)
				{
					m = A->val [A->ncol * i + scol] / A->val [A->ncol * srow + scol];
					for (uint32_t j = scol; j < A->ncol; j++)
						A->val [A->ncol * i + j] -= m * A->val [A->ncol * srow + j];
					b->val [i] -= m * b->val [srow];
				}

			srow++;
			scol++;
		}

	/* Back substitution. */
	for (int32_t i = A->nrow - 1; i >= 0; i--)
		{
			b->val [i] /= A->val [A->ncol * i + i];
			for (uint32_t j = i + 1; j < A->ncol; j++)
				b->val [i] -= b->val [j] * A->val [A->ncol * i + j] / A->val [A->ncol * i + i];
		}

	gettimeofday (&end, NULL);

	std::cout << "single end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;
}

void
replicate (struct dense_mtx *A, struct dense_mtx *b, struct dense_mtx *_A, struct dense_mtx *_b)
{
	for (uint32_t i = 0; i < A->nrow; i++)
		for (uint32_t j = 0; j < A->ncol; j++)
			_A->val [_A->ncol * i + j] = A->val [A->ncol * i + j];

	for (uint32_t i = 0; i < b->nrow; i++)
		_b->val [i] = b->val [i];
}

void
L2norm (struct dense_mtx *A, struct dense_mtx *b, struct dense_mtx *_b)
{
  double diffsq, temp;

	diffsq = 0;

	for (uint32_t i = 0; i < A->nrow; i++)
		{
			temp = 0;
			for (uint32_t j = 0; j < A->ncol; j++)
				temp += A->val [A->ncol * i + j] * _b->val [j];
			diffsq += (b->val [i] - temp) * (b->val [i] - temp);
		}

	std::cout << "L2-norm: " << sqrt (diffsq) << std::endl;
}

