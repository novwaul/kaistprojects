#include "hip/hip_runtime.h"
#include "mmreader.hpp"
#include <time.h>
#include <iostream>
#include <sys/time.h>
#include <unistd.h>
#include <omp.h>
#include <pthread.h>
#include <list>
#include <cmath>

#define WARP 32
#define MAXBLOCK 65535

void validate (struct dense_mtx *C1, struct dense_mtx *C2);

bool
SCsrMatrixfromFile(struct sparse_mtx *A, const char* filePath)
{
    // Check that the file format is matrix market; the only format we can read right now
    // This is not a complete solution, and fails for directories with file names etc...
    // TODO: Should we use boost filesystem?
    std::string strPath( filePath );
    if( strPath.find_last_of( '.' ) != std::string::npos )
    {
        std::string ext = strPath.substr( strPath.find_last_of( '.' ) + 1 );
        if( ext != "mtx" )
        {
            std::cout << "Reading file name error" << std::endl;
            return false;
        }
    }
    else
        return false;

    // Read data from a file on disk into buffers
    // Data is read natively as COO format with the reader
    MatrixMarketReader mm_reader;
    if( mm_reader.MMReadFormat(filePath) )
        return false;

    // JPA: Shouldn't that just be an assertion check? It seems to me that
    // the user have to call clsparseHeaderfromFile before calling this function,
    // otherwise the whole pCsrMatrix will be broken;
    A->nrow = mm_reader.GetNumRows( );
    A->ncol = mm_reader.GetNumCols( );
    A->nnze = mm_reader.GetNumNonZeroes( );

    A->row = (int32_t *)malloc((A->nrow + 1) * sizeof(int32_t));
    A->val = (float *)malloc(A->nnze * sizeof(float));
    A->col = (int32_t *)malloc(A->nnze * sizeof(int32_t));

    if(A->row == NULL || A->col == NULL || A->val == NULL)
    {
        if(A->row == NULL)
            free((void *)A->row);
        if(A->col == NULL)
            free((void *)A->col);
        if(A->val == NULL)
            free((void *)A->val);
        return false;
    }

    //  The following section of code converts the sparse format from COO to CSR
    Coordinate* coords = mm_reader.GetUnsymCoordinates( );

    std::sort( coords, coords + A->nnze, CoordinateCompare );

    int32_t current_row = 1;

    A->row[ 0 ] = 0;

    for (int32_t i = 0; i < A->nnze; i++)
    {
        A->col[ i ] = coords[ i ].y;
        A->val[ i ] = coords[ i ].val;

        while( coords[ i ].x >= current_row )
            A->row[ current_row++ ] = i;
    }

    A->row[ current_row ] = A->nnze;

    while( current_row <= A->nrow )
        A->row[ current_row++ ] = A->nnze;

    return true;
}

void multiply_single(struct sparse_mtx *A, struct dense_mtx *B, struct dense_mtx *C)
{
    C->nrow = A->nrow;
    C->ncol = B->ncol;
    C->val = (float *)calloc(1, C->nrow * C->ncol * sizeof(float));

    if(C->val == NULL)
        return;
    
    for(int32_t i = 0; i < A->nrow; i++)
    {
        int32_t A_col_start = A->row[i];
        int32_t A_col_stop = A->row[i + 1];
        
        for(int32_t j = A_col_start; j < A_col_stop; j++)
        {
            int32_t B_row = A->col[j];

            for(int32_t k = 0; k < B->ncol; k++)
                C->val[i * C->ncol + k] += A->val[j] * B->val[B_row * B->ncol + k];
        }
    }
}

__global__ void
multiply_matrix (int32_t *A_row, int32_t *A_col, float *A_val, float *B, float *C, uint32_t n, uint32_t m, int32_t ofs)
{
	volatile __shared__ float vals [WARP];
	int32_t col_start;
	int32_t col_stop;
	int32_t tx = threadIdx.x;
	int32_t Row = ofs + blockIdx.y;
	int32_t Col = blockIdx.x;

	vals [tx] = 0.0f;

	col_start = A_row [Row];
	col_stop = A_row [Row + 1];

	for (int32_t j = col_start + tx; col_start <= j && j < col_stop; j += WARP)
		{
			float coef = A_val [j];
			float bval = B [A_col [j] * m + Col];
			vals [tx] += bval * coef;
		}

	int32_t size = WARP;
	while (size > 1)
		{
			size /= 2;
			if (tx < size)
				{
					vals [tx] += vals [tx + size];
				}
			else;
		}
	
	if (tx == 0)
		{
			C [Row * m + Col] = vals [0];
		}

	else;
}

void cuda (struct sparse_mtx *A, struct dense_mtx *B, struct dense_mtx *C)
{
	C->nrow = A->nrow;
	C->ncol = B->ncol;
	/* Allocate C value memory */
	C->val = (float *) calloc (1, sizeof (float) * C->nrow * C->ncol);
	
	/* Allocate GPU memory */
	int32_t *Ad_row, *Ad_col;
	float *Ad_val, *Bd, *Cd;
	if (hipMalloc (&Ad_row, sizeof (int32_t) * (A->nrow + 1)) != hipSuccess
			|| hipMalloc (&Ad_col, sizeof (int32_t) * A->nnze) != hipSuccess
			|| hipMalloc (&Ad_val, sizeof (float) * A->nnze) != hipSuccess
			|| hipMalloc (&Bd, sizeof (float) * B->nrow * B->ncol) != hipSuccess
			|| hipMalloc (&Cd, sizeof (float) * C->nrow * C->ncol) != hipSuccess )
	{
		printf ("Memory Allocation Error!\n");
		return;
	}

	/* Send data. */
	if (hipMemcpy (Ad_row, A->row, sizeof (int32_t) * (A->nrow + 1), hipMemcpyHostToDevice) != hipSuccess
			|| hipMemcpy (Ad_col, A->col, sizeof (int32_t) * A->nnze, hipMemcpyHostToDevice) != hipSuccess
			|| hipMemcpy (Ad_val, A->val, sizeof (float) * A->nnze, hipMemcpyHostToDevice) != hipSuccess
			|| hipMemcpy (Bd, B->val, sizeof (float) * B->nrow * B->ncol, hipMemcpyHostToDevice) != hipSuccess)
	{
		printf ("Memcpy Error: Host to Device\n");
		return;
	}

	int32_t ofs = 0;
	int32_t size = C->nrow;
	while (size > MAXBLOCK)
		{
			dim3 dimGrid (C->ncol, MAXBLOCK, 1);
			
			multiply_matrix<<<dimGrid, WARP>>> (Ad_row, Ad_col, Ad_val, Bd, Cd, C->nrow, C->ncol, ofs);

			if (hipSuccess != hipGetLastError ())
				{
					printf ("Kernel Error\n");
					return;
				}

			size -= MAXBLOCK;
			ofs += MAXBLOCK;
		}

	dim3 dimGrid (C->ncol, size, 1);
	multiply_matrix<<<dimGrid, WARP>>> (Ad_row, Ad_col, Ad_val, Bd, Cd, C->nrow, C->ncol, ofs);

	if (hipSuccess != hipGetLastError ())
		{
			printf ("Kernel Error\n");
			return;
		}

	/* Get data. */
	if (hipMemcpy (C->val, Cd, sizeof (float) * C->nrow * C->ncol, hipMemcpyDeviceToHost) != hipSuccess)
		{
			printf ("Memcpy Error: Device to Host\n");
			return;
		}

	/* Deallocate GPU memory. */
	hipFree (Ad_row);
	hipFree (Ad_col);
	hipFree (Ad_val);
	hipFree (Bd);
	hipFree (Cd);
}

uint64_t GetTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv,NULL);
    return tv.tv_sec*(uint64_t)1000000+tv.tv_usec;
}

int main(int argc, char **argv)
{
    struct sparse_mtx A;
    if(!SCsrMatrixfromFile(&A, argv[1]))
    {
        std::cout << "read failed." << std::endl;
        return 0;
    }
    std::cout << "Matrix: " << argv[1] << std::endl;

    struct dense_mtx B;
    B.nrow = A.ncol;
    B.ncol = atoi(argv[2]);

    if((int) B.ncol < 0)
    {
        free(A.row);
        free(A.col);
        free(A.val);
        std::cerr << "Invalid argument for the number of columns of B." << std::endl;
    }

    B.val = (float *)malloc(sizeof(float) * B.nrow * B.ncol);

    srand((unsigned int)time(NULL));
    for(int i = 0; i < B.nrow; i++)
    {
        for(int j = 0; j < B.ncol; j++)
        {
            B.val[B.ncol * i + j] = ((float)rand()/(float)(RAND_MAX)) * ((rand() % 2) ? 1.0f : -1.0f);
        }
    }

    struct dense_mtx C1, C2;
    C1.val = C2.val = NULL;
		
	struct timeval start, end;

		/* Single. */
		std::cout << "single start..." << std::endl;
    gettimeofday (&start, NULL);
    multiply_single(&A, &B, &C1);
    gettimeofday (&end, NULL);
    std::cout << "single end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;

		/* GPU. */
		std::cout << "cuda start..." << std::endl;
    gettimeofday (&start, NULL);
    cuda (&A, &B, &C2);
    gettimeofday (&end, NULL);
    std::cout << "cuda end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;

		validate (&C1, &C2);

    free(A.row);
    free(A.col);
    free(A.val);
    free(B.val);
    if (C1.val != NULL)
      free(C1.val);
 		if (C2.val != NULL)
      free(C2.val); 
   
    return 0;
}

void
validate (struct dense_mtx *C1, struct dense_mtx *C2)
{
	float diff;
	float L2norm = 0.000000f;
	for (int32_t i = 0; i < C1->nrow; i++)
		for (int32_t j = 0; j < C1->ncol; j++)
			{
				float c1val = C1->val [i * C1->ncol + j];
				float c2val = C2->val [i * C2->ncol + j];
				diff = c1val - c2val;
				while (c1val >= 1 || c1val <= -1 || c2val >= 1 || c2val <= -1)
					{
						c1val /= 10;
						c2val /= 10;
						diff /= 10;
					}
				L2norm += diff * diff;
			}
	printf ("L2norm: %f\n", sqrt (L2norm));
}
