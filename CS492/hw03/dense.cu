
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <ctime>
#include <cfloat>
#include <omp.h>
#include <sys/time.h>
#include <cmath>

#define MAX_THREADS 6

struct dense_mtx
{
    uint32_t nrow; /* Number of rows */
    uint32_t ncol; /* Number of columns */
    double *val;   /* Value of this matrix. */
};

void single (struct dense_mtx *A, struct dense_mtx *B, struct dense_mtx *C);
void cuda (struct dense_mtx *A, struct dense_mtx *B, struct dense_mtx *C);

int 
main (int argc, char** argv)
{
	uint32_t n = atoi (argv[1]);
	uint32_t total_size = n * n;

  struct dense_mtx A, B, C1, C2;

	
	A.ncol = A.nrow = n;
	A.val = (double *) malloc (total_size * sizeof (double));

	B.ncol = B.nrow = n;
	B.val = (double *) malloc (total_size * sizeof (double));

	C1.ncol = C1.nrow = n;
	C2.ncol = C2.nrow = n;

	srand48 (time (NULL));

	for (int i = 0; i < total_size; i++)
	  {
			A.val[i] = drand48 ();
			B.val[i] = drand48 ();
		}
  
	C1.val = (double *) calloc (1, total_size * sizeof (double));
	single (&A, &B, &C1);
  
	C2.val = (double *) calloc (1, total_size * sizeof (double));
	cuda (&A, &B, &C2);
  
	double L2norm = 0, diff;
	for (int i = 0; i < total_size; i++)
		{
			diff = (C1.val[i] - C2.val[i]);
			L2norm += diff * diff;
	 	}

	std::cout << "L2norm: " << sqrt (L2norm) << std::endl;

	free (C1.val);
	free (C2.val);
	free (A.val);
	free (B.val);

	return 0;
}

void
single (struct dense_mtx *A, struct dense_mtx *B, struct dense_mtx *C)
{
	struct timeval start, end;
	uint32_t Block = 32;

  std::cout << "single start..." << std::endl;

	gettimeofday (&start, NULL);

	for (uint32_t i = 0; i < C->nrow; i += Block)
		for (uint32_t j = 0; j < C->ncol; j += Block)
			for (uint32_t k = 0; k < C->ncol; k += Block)
				for (uint32_t _i = i; _i < (((i + Block) < C->nrow) ? i + Block : C->nrow); _i++)
					for (uint32_t _j = j; _j < (((j + Block) < C->ncol) ? j + Block : C->ncol); _j++)
						for (uint32_t _k = k; _k < (((k + Block) < C->ncol) ? k + Block : C->ncol); _k++)
							C->val [C->nrow * _i + _j] += A->val [A->nrow * _i + _k] * B->val [B->nrow * _k + _j];
							
	gettimeofday (&end, NULL);

	std::cout << "single end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;
}

	__global__ void
matrix_multiply (double *A, double *B, double *C, uint32_t n)
{
	uint32_t Block = 16;

  __shared__ double TileA[16][16];
	__shared__ double TileB[16][16];

	uint32_t tx = threadIdx.x;
	uint32_t ty = threadIdx.y;
	uint32_t Row = Block * blockIdx.y + ty;
	uint32_t Col = Block * blockIdx.x + tx;

	double val = 0;

	for (uint32_t i = 0; i < (n + Block - 1) / Block; i++)
		{
			/* Fetch data to shared memeory. */
			if ((i * Block + tx) < n && Row < n)
				TileA [ty][tx] = A [Row * n + i * Block + tx];
			else
				TileA [ty][tx] = 0;

			if ((i * Block + ty) < n && Col < n)
				TileB [ty][tx] = B [(i * Block + ty) * n + Col];
			else
				TileB [ty][tx] = 0;

			__syncthreads ();
			for (uint32_t k = 0; k < Block; k++)
				val += TileA [ty][k] * TileB [k][tx];
			__syncthreads ();
		}
	if (Row < n && Col < n)
	  C [Row * n + Col] = val;
}

void
cuda (struct dense_mtx *A, struct dense_mtx *B, struct dense_mtx *C)
{
	struct timeval start, end;
	uint32_t Block = 16;
	uint32_t size;
	double *A_val, *B_val, *C_val;

  std::cout << "cuda start..." << std::endl;

	gettimeofday (&start, NULL);

  dim3 dimGrid ((A->nrow + Block - 1) / Block, (A->ncol + Block - 1) / Block, 1);
	dim3 dimBlock (Block, Block, 1);

	size = A->nrow * A->ncol * sizeof (double);
	
	hipMalloc (&A_val, size);
	hipMalloc (&B_val, size);
	hipMalloc (&C_val, size);

	hipMemcpy (A_val, A->val, size, hipMemcpyHostToDevice);
	hipMemcpy (B_val, B->val, size, hipMemcpyHostToDevice);

	matrix_multiply<<<dimGrid, dimBlock>>>(A_val, B_val, C_val, A->nrow);

	hipMemcpy (C->val, C_val, size, hipMemcpyDeviceToHost);

	hipFree (A_val);
	hipFree (B_val);
	hipFree (C_val);

	gettimeofday (&end, NULL);

	std::cout << "cuda end: " << (double) (end.tv_usec - start.tv_usec) / 1000000 
		+ (double) (end.tv_sec - start.tv_sec) << " s." << std::endl;
}
